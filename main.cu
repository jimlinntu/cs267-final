// Adapted from
// https://github.com/deeperlearning/professional-cuda-c-programming/blob/master/solutions/chapter08/cusparse-matrix-matrix.cu
// https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuSPARSE/spmm_csr
//

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>

#include <assert.h>
#include <iostream>

struct Algo{
    void spmm();
    void sddmm();
    void sddmm_spmm();
};

struct MatrixGenerator{
    void generate_sparse_csr(int, int);
    void generate_dense(int, int);
};

struct CusparseAlgo{
    void spmm(hipsparseHandle_t &handle,
        hipsparseSpMatDescr_t &S,
        hipsparseDnMatDescr_t &A, hipsparseDnMatDescr_t &C);
    void sddmm();
    void sddmm_spmm();
};

// ---------------------------
void Algo::spmm(){
}

void Algo::sddmm(){
}

void Algo::sddmm_spmm(){
}

void CusparseAlgo::spmm(
        hipsparseHandle_t &handle,
        hipsparseSpMatDescr_t &S,
        hipsparseDnMatDescr_t &A, hipsparseDnMatDescr_t &C){

    double alpha = 1.0, beta = 0.;
    size_t bufsize = 0;
    hipsparseSpMM_bufferSize(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, S, A, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT,
            &bufsize);
    void *dbuf = NULL;
    assert(hipMalloc(&dbuf, bufsize) == hipSuccess);
    assert(hipsparseSpMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, S, A, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dbuf) == hipSuccess);
    assert(hipFree(dbuf) == hipSuccess);
}

void CusparseAlgo::sddmm(){
}

void CusparseAlgo::sddmm_spmm(){
}

void MatrixGenerator::generate_sparse_csr(int num_rows, int num_cols){
}

void MatrixGenerator::generate_dense(int num_rows, int num_cols){
}

// forward declaration
struct HostDenseMat;
struct HostSparseMat;

struct DeviceDenseMat;
struct DeviceSparseMat;

struct HostDenseMat{
    int num_rows, num_cols;
    double *vals;
    bool to_delete;
    HostDenseMat(int num_rows_, int num_cols_, double *vals_);
    ~HostDenseMat();
    void to_device(DeviceDenseMat &d);
    friend std::ostream& operator<<(std::ostream &os, const HostDenseMat &obj);
};

struct DeviceDenseMat{
    int num_rows, num_cols;
    double *vals;
    DeviceDenseMat() = default;
    ~DeviceDenseMat();

    void get_cusparse_descriptor(hipsparseDnMatDescr_t &mat);
    void copy_to_host(HostDenseMat &h);
};

struct DeviceSparseMat{
    int num_rows, num_cols;
    int nnz;
    int *offsets;
    int *cols;
    double *vals;

    DeviceSparseMat() = default;

    DeviceSparseMat(
            int num_rows_, int num_cols_, int nnz_,
            int *offsets_, int *cols_, double *vals_)
        :num_rows(num_rows_), num_cols(num_cols_), nnz(nnz_),
         offsets(offsets_), cols(cols_), vals(vals_){
    }
    ~DeviceSparseMat(){
        assert(hipFree(offsets) == hipSuccess);
        assert(hipFree(cols) == hipSuccess);
        assert(hipFree(vals) == hipSuccess);
    }
    void get_cusparse_descriptor(
        hipsparseSpMatDescr_t &mat){

        hipsparseCreateCsr(&mat, num_rows, num_cols, nnz,
                          offsets, cols, vals,
                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    }
};

DeviceDenseMat::~DeviceDenseMat(){
    assert(hipFree(vals) == hipSuccess);
}
void DeviceDenseMat::get_cusparse_descriptor(hipsparseDnMatDescr_t &mat){
    assert(hipsparseCreateDnMat(&mat, num_rows, num_cols, num_cols, vals, HIP_R_64F, HIPSPARSE_ORDER_ROW) == hipSuccess);
}
void DeviceDenseMat::copy_to_host(HostDenseMat &h){
    assert(h.num_rows == num_rows);
    assert(h.num_cols == num_cols);
    assert(hipMemcpy(h.vals, vals, num_rows * num_cols * sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);
}

std::ostream& operator<<(std::ostream &os, const HostDenseMat &obj){
    for(int i = 0; i < obj.num_rows; ++i){
        for(int j = 0; j < obj.num_cols; ++j){
            os << obj.vals[i*obj.num_cols + j] << "\t";
        }
        os << "\n";
    }
    return os;
}


struct HostSparseMat{
    int num_rows, num_cols;
    int nnz;
    int *offsets;
    int *cols;
    double *vals;
    bool to_delete;
    HostSparseMat(
            int num_rows_, int num_cols_, int nnz_,
            int *offsets_, int *cols_, double *vals_)
        :num_rows(num_rows_), num_cols(num_cols_), nnz(nnz_),
         offsets(offsets_), cols(cols_), vals(vals_), to_delete(false){
    }
    ~HostSparseMat(){
        if(!to_delete) return;

        delete offsets;
        delete cols;
        delete vals;
    }
    void to_device(DeviceSparseMat &d){
        d.num_rows = num_rows;
        d.num_cols = num_cols;
        d.nnz = nnz;

        // malloc
        assert(hipMalloc(&d.offsets, (num_rows+1) * sizeof(int)) == hipSuccess);
        assert(hipMalloc(&d.cols, nnz * sizeof(int)) == hipSuccess);
        assert(hipMalloc(&d.vals, nnz * sizeof(double)) == hipSuccess);

        // copy
        assert(hipMemcpy(d.offsets, offsets, (num_rows+1) * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
        assert(hipMemcpy(d.cols, cols, nnz * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
        assert(hipMemcpy(d.vals, vals, nnz * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
    }
};

HostDenseMat::HostDenseMat(int num_rows_, int num_cols_, double *vals_)
        :num_rows(num_rows_), num_cols(num_cols_), vals(vals_), to_delete(false){

}

HostDenseMat::~HostDenseMat(){
    if(!to_delete) return;
    delete vals;
}

void HostDenseMat::to_device(DeviceDenseMat &d){
    d.num_rows = num_rows;
    d.num_cols = num_cols;

    assert(hipMalloc(&d.vals, num_rows * num_cols * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d.vals, vals, num_rows * num_cols * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
}


void test_spmm(){
    // C = S @ A
    int S_num_rows = 4, S_num_cols = 4;
    int S_nnz = 9;
    int hS_offsets[] = {0, 3, 4, 7, 9};
    int hS_cols[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};
    double hS_vals[] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0};

    int A_num_rows = S_num_cols, A_num_cols = 3;
    double hA[] = {1.0f,  2.0f,  3.0f,  4.0f,
              5.0f,  6.0f,  7.0f,  8.0f,
              9.0f, 10.0f, 11.0f, 12.0f };

    int C_num_rows = S_num_rows, C_num_cols = A_num_cols;
    double hC[4*3] = {0};


    HostSparseMat S(S_num_rows, S_num_cols, S_nnz,
                  hS_offsets, hS_cols, hS_vals);

    HostDenseMat A(A_num_rows, A_num_cols, hA);

    HostDenseMat C(C_num_rows, C_num_cols, hC);


    DeviceSparseMat dS;
    DeviceDenseMat dA, dC;

    S.to_device(dS);
    A.to_device(dA);
    C.to_device(dC);

    // Initialize environment
    {
        hipsparseHandle_t handle = NULL;
        assert(hipsparseCreate(&handle) == hipSuccess);

        hipsparseSpMatDescr_t S_des;
        hipsparseDnMatDescr_t A_des, C_des;

        // Convert them to cusparse descriptors
        dS.get_cusparse_descriptor(S_des);
        dA.get_cusparse_descriptor(A_des);
        dC.get_cusparse_descriptor(C_des);

        CusparseAlgo cualgo;

        // Execute spmm algorithm
        cualgo.spmm(handle, S_des, A_des, C_des);

        // copy back
        dC.copy_to_host(C);

        // Print the result
        std::cout << C;

        assert(hipsparseDestroy(handle) == hipSuccess);
        assert(hipsparseDestroySpMat(S_des) == hipSuccess);
        assert(hipsparseDestroyDnMat(A_des) == hipSuccess);
        assert(hipsparseDestroyDnMat(C_des) == hipSuccess);
    }
}

int main(){
    test_spmm();
    return 0;
}
