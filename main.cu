// Adapted from
// https://github.com/deeperlearning/professional-cuda-c-programming/blob/master/solutions/chapter08/cusparse-matrix-matrix.cu
// https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuSPARSE/spmm_csr
//

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <iomanip>
#include <assert.h>
#include <iostream>
#include <cmath>
#include <limits> 

#define TILE_WIDTH 4

// forward declaration
struct HostDenseMat;
struct DeviceDenseMat;

struct HostSparseMat;
struct DeviceSparseMat;

struct HostDenseMat{
    int num_rows, num_cols;
    double *vals;
    bool to_delete;
    HostDenseMat(int num_rows_, int num_cols_, double *vals_);
    ~HostDenseMat();
    void to_device(DeviceDenseMat &d);
    friend std::ostream& operator<<(std::ostream &os, const HostDenseMat &obj);
    // overload set
    double & operator [](int i) {return vals[i];}
    // overload get
    double operator [](int i) const {return vals[i];};
    // overload comparison
    bool operator == (const HostDenseMat& m2) {
        if(num_rows != m2.num_rows) return false;
        if(num_cols != m2.num_cols) return false;
        double epsilon = 1e-4; // std::numeric_limits<double>::epsilon();
        for(int i = 0; i < num_rows; i++)
            for(int j = 0; j < num_cols; j++)
                if(std::fabs(vals[i*num_cols+j] - m2.vals[i*num_cols+j]) > epsilon)
                    return false;
        return true;
    }
    bool operator != (const HostDenseMat& m2) {
        return !(*this == m2);
    }
};

struct DeviceDenseMat{
    int num_rows, num_cols;
    double *vals;
    DeviceDenseMat() = default;
    ~DeviceDenseMat();

    void get_cusparse_descriptor(hipsparseDnMatDescr_t &mat);
    void copy_to_host(HostDenseMat &h);
    // overload set
    __device__ double & operator [](int i) {return vals[i];}
    // overload get
    __device__ double operator [](int i) const {return vals[i];};
};

struct HostSparseMat{
    int num_rows, num_cols;
    int nnz;
    int *offsets;
    int *cols;
    double *vals;
    bool to_delete;
    HostSparseMat(
            int num_rows_, int num_cols_, int nnz_,
            int *offsets_, int *cols_, double *vals_);
    ~HostSparseMat();
    void to_device(DeviceSparseMat &d);
};

struct DeviceSparseMat{
    int num_rows, num_cols;
    int nnz;
    int *offsets;
    int *cols;
    double *vals;

    DeviceSparseMat() = default;
    DeviceSparseMat(int num_rows_, int num_cols_, int nnz_,
            int *offsets_, int *cols_, double *vals_);
    ~DeviceSparseMat();
    void get_cusparse_descriptor(hipsparseSpMatDescr_t &mat);
};

struct Algo{
    void spmm();
    void sddmm();
    void sddmm_spmm();
    void ddmm_seq(HostDenseMat &, HostDenseMat &, HostDenseMat &);
    void ddmm(HostDenseMat &, HostDenseMat &, HostDenseMat &);
};

struct MatrixGenerator{
    void generate_sparse_csr(int, int);
    void generate_dense(int, int, double**);
};

struct CusparseAlgo{
    void spmm(hipsparseHandle_t &handle,
        hipsparseSpMatDescr_t &S,
        hipsparseDnMatDescr_t &A, hipsparseDnMatDescr_t &C);
    void sddmm();
    void sddmm_spmm();
};


// ---------------------------
void Algo::spmm(){
}

void Algo::sddmm(){
}

void Algo::sddmm_spmm(){
}

void Algo::ddmm_seq(HostDenseMat &A, HostDenseMat &B, HostDenseMat &C){
    int A_num_rows = A.num_rows, A_num_cols = A.num_cols;
    int B_num_rows = B.num_rows, B_num_cols = B.num_cols;
    assert(A_num_cols == B_num_rows);

    for(int i = 0; i < A_num_rows; i++) {
        for(int j = 0; j < B_num_cols; j++) {
            C[i*B_num_cols+j] = 0.0;
            for(int k = 0; k < A_num_cols; k++) {
                C[i*B_num_cols+j] += A[i*A_num_cols+k] * B[k*B_num_cols+j];
            }
        }
    }
}


__global__ void ddmm_kernel(double* A, double* B, double* C, int A_h, int A_w, int B_h, int B_w) {
    __shared__ double As[TILE_WIDTH][TILE_WIDTH];
    __shared__ double Bs[TILE_WIDTH][TILE_WIDTH];

    int gx_C = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int gy_C = blockIdx.y * TILE_WIDTH + threadIdx.y;
    double value_C = 0.0;

    for(int m = 0; m < (A_w+TILE_WIDTH-1) / TILE_WIDTH; m++) {
        int lx_A = threadIdx.x;
        int ly_A = threadIdx.y;
        int gx_A = m * TILE_WIDTH + threadIdx.x;
        int gy_A = gy_C;
        if(gy_A < A_h && gx_A < A_w)
            As[ly_A][lx_A] = A[gy_A * A_w + gx_A];
        else // out of range
            As[ly_A][lx_A] = 0.0;

        int lx_B = threadIdx.x;
        int ly_B = threadIdx.y;
        int gx_B = gx_C;
        int gy_B = m * TILE_WIDTH + threadIdx.y;
        if(gy_B < B_h && gx_B < B_w)
            Bs[ly_B][lx_B] = B[gy_B * B_w + gx_B];
        else
            Bs[ly_B][lx_B] = 0.0;
        
        // printf("As[%d][%d]=%f\n", ly_A, lx_A, As[ly_A][lx_A]);
        // printf("Bs[%d][%d]=%f\n", ly_B, lx_B, Bs[ly_A][lx_A]);
        __syncthreads();

        for(int k = 0; k < TILE_WIDTH; k++)
            value_C += As[ly_A][k] * Bs[k][lx_B];

        __syncthreads();

    }

    // printf("C[%d][%d]=%f\n", gy_C, gx_C, value_C);
    if(gy_C < A_h && gx_C < B_w) // make sure in range
        C[gy_C * B_w + gx_C] = value_C;
    
}

void Algo::ddmm(HostDenseMat &A, HostDenseMat &B, HostDenseMat &C){
    DeviceDenseMat d_A, d_B, d_C;
    A.to_device(d_A);
    B.to_device(d_B);
    C.to_device(d_C);

    int A_h = A.num_rows, A_w = A.num_cols, B_h = B.num_rows, B_w = B.num_cols;
    dim3 dimGrid((B_w+TILE_WIDTH-1)/TILE_WIDTH, (A_h+TILE_WIDTH-1)/TILE_WIDTH); // need to solve non-divisible
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    ddmm_kernel<<<dimGrid, dimBlock>>>(d_A.vals, d_B.vals, d_C.vals, A_h, A_w, B_h, B_w);

    d_C.copy_to_host(C);
}

void CusparseAlgo::spmm(
        hipsparseHandle_t &handle,
        hipsparseSpMatDescr_t &S,
        hipsparseDnMatDescr_t &A, hipsparseDnMatDescr_t &C){

    double alpha = 1.0, beta = 0.;
    size_t bufsize = 0;
    hipsparseSpMM_bufferSize(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, S, A, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT,
            &bufsize);
    void *dbuf = NULL;
    assert(hipMalloc(&dbuf, bufsize) == hipSuccess);
    assert(hipsparseSpMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, S, A, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dbuf) == hipSuccess);
    assert(hipFree(dbuf) == hipSuccess);
}

void CusparseAlgo::sddmm(){
}

void CusparseAlgo::sddmm_spmm(){
}

void MatrixGenerator::generate_sparse_csr(int num_rows_, int num_cols_){
}

void MatrixGenerator::generate_dense(int num_rows_, int num_cols_, double** vals){
    *vals = new double[num_rows_ * num_cols_];
    for(int i = 0; i < num_rows_; i++)
        for(int j = 0; j < num_cols_; j++)
            (*vals)[i*num_cols_+j] = ((double)rand()/(double)RAND_MAX);
}

// ==
HostDenseMat::HostDenseMat(int num_rows_, int num_cols_, double* vals_)
        :num_rows(num_rows_), num_cols(num_cols_), vals(vals_), to_delete(false){
}

HostDenseMat::~HostDenseMat(){
    if(!to_delete) return;
    delete vals;
}

void HostDenseMat::to_device(DeviceDenseMat &d){
    d.num_rows = num_rows;
    d.num_cols = num_cols;

    assert(hipMalloc(&d.vals, num_rows * num_cols * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d.vals, vals, num_rows * num_cols * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
}

std::ostream& operator<<(std::ostream &os, const HostDenseMat &obj){
    for(int i = 0; i < obj.num_rows; ++i){
        for(int j = 0; j < obj.num_cols; ++j){
            os << std::right << std::setw(6) << std::setprecision(4) << obj.vals[i*obj.num_cols + j] << "\t";
        }
        os << "\n";
    }
    return os;
}


DeviceDenseMat::~DeviceDenseMat(){
    assert(hipFree(vals) == hipSuccess);
}
void DeviceDenseMat::get_cusparse_descriptor(hipsparseDnMatDescr_t &mat){
    assert(hipsparseCreateDnMat(&mat, num_rows, num_cols, num_cols, vals, HIP_R_64F, HIPSPARSE_ORDER_ROW) == hipSuccess);
}
void DeviceDenseMat::copy_to_host(HostDenseMat &h){
    assert(h.num_rows == num_rows);
    assert(h.num_cols == num_cols);
    assert(hipMemcpy(h.vals, vals, num_rows * num_cols * sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);
}


HostSparseMat::HostSparseMat(
            int num_rows_, int num_cols_, int nnz_,
            int *offsets_, int *cols_, double *vals_)
    :num_rows(num_rows_), num_cols(num_cols_), nnz(nnz_),
     offsets(offsets_), cols(cols_), vals(vals_), to_delete(false){
}

HostSparseMat::~HostSparseMat(){
    if(!to_delete) return;

    delete offsets;
    delete cols;
    delete vals;
}

void HostSparseMat::to_device(DeviceSparseMat &d){
    d.num_rows = num_rows;
    d.num_cols = num_cols;
    d.nnz = nnz;

    // malloc
    assert(hipMalloc(&d.offsets, (num_rows+1) * sizeof(int)) == hipSuccess);
    assert(hipMalloc(&d.cols, nnz * sizeof(int)) == hipSuccess);
    assert(hipMalloc(&d.vals, nnz * sizeof(double)) == hipSuccess);

    // copy
    assert(hipMemcpy(d.offsets, offsets, (num_rows+1) * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(d.cols, cols, nnz * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(d.vals, vals, nnz * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
}

DeviceSparseMat::DeviceSparseMat(
        int num_rows_, int num_cols_, int nnz_,
        int *offsets_, int *cols_, double *vals_)
    :num_rows(num_rows_), num_cols(num_cols_), nnz(nnz_),
     offsets(offsets_), cols(cols_), vals(vals_){
}

DeviceSparseMat::~DeviceSparseMat(){
    assert(hipFree(offsets) == hipSuccess);
    assert(hipFree(cols) == hipSuccess);
    assert(hipFree(vals) == hipSuccess);
}

void DeviceSparseMat::get_cusparse_descriptor(
    hipsparseSpMatDescr_t &mat){

    hipsparseCreateCsr(&mat, num_rows, num_cols, nnz,
                      offsets, cols, vals,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
}

void test_spmm(){
    // C = S @ A
    int S_num_rows = 4, S_num_cols = 4;
    int S_nnz = 9;
    int hS_offsets[] = {0, 3, 4, 7, 9};
    int hS_cols[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};
    double hS_vals[] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0};

    int A_num_rows = S_num_cols, A_num_cols = 3;
    double hA[] = {1.0f,  2.0f,  3.0f,  4.0f,
              5.0f,  6.0f,  7.0f,  8.0f,
              9.0f, 10.0f, 11.0f, 12.0f };

    int C_num_rows = S_num_rows, C_num_cols = A_num_cols;
    double hC[4*3] = {0};


    HostSparseMat S(S_num_rows, S_num_cols, S_nnz,
                  hS_offsets, hS_cols, hS_vals);

    HostDenseMat A(A_num_rows, A_num_cols, hA);

    HostDenseMat C(C_num_rows, C_num_cols, hC);


    DeviceSparseMat dS;
    DeviceDenseMat dA, dC;

    S.to_device(dS);
    A.to_device(dA);
    C.to_device(dC);

    // Initialize environment
    {
        hipsparseHandle_t handle = NULL;
        assert(hipsparseCreate(&handle) == hipSuccess);

        hipsparseSpMatDescr_t S_des;
        hipsparseDnMatDescr_t A_des, C_des;

        // Convert them to cusparse descriptors
        dS.get_cusparse_descriptor(S_des);
        dA.get_cusparse_descriptor(A_des);
        dC.get_cusparse_descriptor(C_des);

        CusparseAlgo cualgo;

        // Execute spmm algorithm
        cualgo.spmm(handle, S_des, A_des, C_des);

        // copy back
        dC.copy_to_host(C);

        // Print the result
        std::cout << C;

        assert(hipsparseDestroy(handle) == hipSuccess);
        assert(hipsparseDestroySpMat(S_des) == hipSuccess);
        assert(hipsparseDestroyDnMat(A_des) == hipSuccess);
        assert(hipsparseDestroyDnMat(C_des) == hipSuccess);
    }
}

void test_ddmm() {
    MatrixGenerator mg;
    Algo alg;
    int A_hs[] = {4, 4, 4, 3, 13};
    int A_ws[] = {4, 8, 16, 1, 5};
    int B_hs[] = {4, 8, 16, 1, 5};
    int B_ws[] = {4, 4, 8, 3, 11};

    for(int i = 0; i < 4; i++) {
        std::cout << "Iteration " << i << ":" << std::endl;
        // matrix A
        int A_num_rows = A_hs[i], A_num_cols = A_ws[i];
        double* A_vals = NULL;
        mg.generate_dense(A_num_rows, A_num_cols, &A_vals);
        HostDenseMat A(A_num_rows, A_num_cols, A_vals);

        // matrix B
        int B_num_rows = B_hs[i], B_num_cols = B_ws[i];
        double* B_vals = NULL;
        mg.generate_dense(B_num_rows, B_num_cols, &B_vals);
        HostDenseMat B(B_num_rows, B_num_cols, B_vals);

        // matrix C
        int C_num_rows = A_num_rows, C_num_cols = B_num_cols;
        double* C_vals = NULL;
        mg.generate_dense(C_num_rows, C_num_cols, &C_vals);
        HostDenseMat C(C_num_rows, C_num_cols, C_vals);

        alg.ddmm_seq(A, B, C);
        std::cout << "Sequential DDMM:" << std::endl;
        std::cout << C;
        // matrix D
        int D_num_rows = A_num_rows, D_num_cols = B_num_cols;
        double* D_vals = NULL;
        mg.generate_dense(D_num_rows, D_num_cols, &D_vals);
        HostDenseMat D(D_num_rows, D_num_cols, D_vals);

        alg.ddmm(A, B, D);
        std::cout << "Blocked DDMM:" << std::endl;
        std::cout << D;
        assert(C == D);
    }
}

int main(){
    // test_spmm();
    test_ddmm();
    return 0;
}
