#include "hip/hip_runtime.h"
#include "../include/benchmark.cuh"

double avg(std::vector<double> &v){
    double a = 0;
    for(double &e: v) a += e;
    return a / (double)v.size();
}

void Benchmarker::benchmark_sddmm(BenchmarkResult &bresult){
    const int S_num_rows = 8023;
    const int A_num_cols = 1049;

    Algo algo;
    CusparseAlgo cualgo;

    MatrixGenerator mg;

    std::map<std::string, std::vector<double>> m;
    clock_t start, end;

    for(int i = 0; i < NUMEXPS; ++i){
        int S_nnz;
        int *S_offsets;
        int *S_cols;
        double *S_vals;
        mg.generate_binary_sparse_csr(S_num_rows, S_num_rows, S_nnz, &S_offsets, &S_cols, &S_vals);

        double *A_vals;
        mg.generate_dense(S_num_rows, A_num_cols, &A_vals);

        int *C_offsets = new int[S_num_rows+1];
        int *C_cols = new int[S_nnz];
        double *C_vals = new double[S_nnz];
        double *C_vals_cusparse = new double[S_nnz];

        memcpy(C_offsets, S_offsets, (S_num_rows+1) * sizeof(int));
        memcpy(C_cols, S_cols, (S_nnz) * sizeof(int));

        HostSparseMat S(S_num_rows, S_num_rows, S_nnz, S_offsets, S_cols, S_vals, true);
        HostDenseMat A(S_num_rows, A_num_cols, A_vals, true);
        HostSparseMat C(S_num_rows, S_num_rows, S_nnz, C_offsets, C_cols, C_vals, true);

        start = clock();
        algo.sddmm(S, A, C);
        end = clock();
        m["sddmm"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        algo.sddmm_block_over_nnz(S, A, C);
        end = clock();
        m["sddmm_block_over_nnz_wo_shm"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        algo.sddmm_block_over_nnz_but_in_same_row(S, A, C);
        end = clock();
        m["sddmm_block_over_nnz_but_in_same_row"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        algo.sddmm_launch_kernel_as_dense_matrix(S, A, C);
        end = clock();
        m["sddmm_launch_kernel_as_dense_matrix"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        algo.sddmm_block_over_nnz_if_same_row_use_shm(S, A, C);
        end = clock();
        m["sddmm_block_over_nnz_if_same_row_use_shm"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        algo.sddmm_dynamic_parallelism(S, A, C);
        end = clock();
        m["sddmm_dynamic_parallelism"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        cualgo.sddmm(S, A, C);
        end = clock();
        m["cusparsesddmm"].push_back((double)(end - start) / CLOCKS_PER_SEC);
    }

    bresult.result["sddmm"] = avg(m["sddmm"]);
    bresult.result["sddmm_block_over_nnz_wo_shm"] = avg(m["sddmm_block_over_nnz_wo_shm"]);
    bresult.result["sddmm_block_over_nnz_but_in_same_row"] = avg(m["sddmm_block_over_nnz_but_in_same_row"]);
    bresult.result["sddmm_launch_kernel_as_dense_matrix"] = avg(m["sddmm_launch_kernel_as_dense_matrix"]);
    bresult.result["sddmm_block_over_nnz_if_same_row_use_shm"] = avg(m["sddmm_block_over_nnz_if_same_row_use_shm"]);
    bresult.result["sddmm_dynamic_parallelism"] = avg(m["sddmm_dynamic_parallelism"]);
    bresult.result["cusparsesddmm"] = avg(m["cusparsesddmm"]);
}

void Benchmarker::benchmark_spmm(BenchmarkResult &bresult){
    const int S_num_rows = 8023;
    const int S_num_cols = 3928;
    const int A_num_rows = S_num_cols;
    const int A_num_cols = 1049;

    Algo algo;
    CusparseAlgo cualgo;

    MatrixGenerator mg;

    std::map<std::string, std::vector<double>> m;
    clock_t start, end;

    for(int i = 0; i < NUMEXPS; ++i){
        int S_nnz;
        int *S_offsets;
        int *S_cols;
        double *S_vals;
        mg.generate_sparse_csr(S_num_rows, S_num_cols, S_nnz, &S_offsets, &S_cols, &S_vals);

        double *A_vals;
        mg.generate_dense(A_num_rows, A_num_cols, &A_vals);

        // Create the output dense matrix
        double *C_vals = new double[S_num_rows * A_num_cols];

        HostSparseMat S(S_num_rows, S_num_cols, S_nnz, S_offsets, S_cols, S_vals, true);
        HostDenseMat A(A_num_rows, A_num_cols, A_vals, true);
        HostDenseMat C(S_num_rows, A_num_cols, C_vals, true);

        start = clock();
        algo.spmm(S, A, C);
        end = clock();
        m["spmm_shm"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        algo.spmm_no_shm(S, A, C);
        end = clock();
        m["spmm_no_shm"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        start = clock();
        cualgo.spmm(S, A, C);
        end = clock();
        m["cusparsespmm"].push_back((double)(end - start) / CLOCKS_PER_SEC);
    }

    bresult.result["spmm_shm"] = avg(m["spmm_shm"]);
    bresult.result["spmm_no_shm"] = avg(m["spmm_no_shm"]);
    bresult.result["cusparsespmm"] = avg(m["cusparsespmm"]);
}

void Benchmarker::benchmark_sddmm_spmm(BenchmarkResult &bresult){
    const int S_num_rows = 8023;
    const int S_num_cols = S_num_rows;
    const int A_num_rows = S_num_rows;
    const int A_num_cols = 1049;

    Algo algo;
    CusparseAlgo cualgo;

    MatrixGenerator mg;

    std::map<std::string, std::vector<double>> m;
    clock_t start, end;

    for(int i = 0; i < NUMEXPS; ++i){
        int S_nnz;
        int *S_offsets;
        int *S_cols;
        double *S_vals;
        mg.generate_sparse_csr(S_num_rows, S_num_cols, S_nnz, &S_offsets, &S_cols, &S_vals);

        double *A_vals;
        mg.generate_dense(A_num_rows, A_num_cols, &A_vals);

        // Create the output dense matrix
        double *C_vals = new double[S_num_rows * A_num_cols];

        HostSparseMat S(S_num_rows, S_num_cols, S_nnz, S_offsets, S_cols, S_vals, true);
        HostDenseMat A(A_num_rows, A_num_cols, A_vals, true);
        HostDenseMat C(S_num_rows, A_num_cols, C_vals, true);

        start = clock();
        algo.sddmm_spmm_block_over_sparse_launch_as_dense_matrix(S, A, C);
        end = clock();
        m["sddmm_spmm_block_over_sparse_launch_as_dense_matrix"].push_back((double)(end - start) / CLOCKS_PER_SEC);

        // This one is extremely slow!! (because of duplicate works)
        if(0){
            start = clock();
            algo.sddmm_spmm_block_over_output(S, A, C);
            end = clock();
            m["sddmm_spmm_block_over_output"].push_back((double)(end - start) / CLOCKS_PER_SEC);
        }

        start = clock();
        cualgo.sddmm_spmm(S, A, C);
        end = clock();
        m["cusparse_sddmm_spmm"].push_back((double)(end - start) / CLOCKS_PER_SEC);
    }

    bresult.result["sddmm_spmm_block_over_sparse_launch_as_dense_matrix"] = avg(m["sddmm_spmm_block_over_sparse_launch_as_dense_matrix"]);
    /* bresult.result["sddmm_spmm_block_over_output"] = avg(m["sddmm_spmm_block_over_output"]); */
    bresult.result["cusparse_sddmm_spmm"] = avg(m["cusparse_sddmm_spmm"]);
}

std::ostream& operator<<(std::ostream &os, const BenchmarkResult &obj){
    auto &result = obj.result;
    for(auto it = result.begin(); it != result.end(); ++it){
        std::string expname = it->first;
        double avg_sec = it->second;

        os << expname << " takes " << avg_sec << " seconds\n";
    }
    return os;
}
