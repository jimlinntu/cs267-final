#include "../include/algo.cuh"

/*********************
Function for MatrixGenerator
******************/

void MatrixGenerator::generate_sparse_csr(int num_rows_, int num_cols_, int &nnz, int** offsets, int** cols, double** vals) {
    double *tmp_vals = new double[num_rows_ * num_cols_];
    double epsilon = 1e-4;
    double zero_ratio = 0.7;
    double val;
    nnz = 0;

    for(int i = 0; i < num_rows_; i++)
        for(int j = 0; j < num_cols_; j++) {
            double p = ((double)rand()/(double)RAND_MAX);
            if(p < zero_ratio)
                val = 0.0;
            else
                val = ((double)rand()/(double)RAND_MAX) + epsilon;

            if(val >= epsilon)
                nnz += 1;

            tmp_vals[i * num_cols_ + j] = val;
        }
    
    *vals = new double[nnz];
    *cols = new int[nnz];
    *offsets = new int[num_rows_+1];
    int vals_cursor = 0;
    int cols_cursor = 0;

    // printf("nnz=%d nr=%d nc=%d\n", nnz, num_rows_, num_cols_);

    for(int i = 0; i < num_rows_; i++){
        (*offsets)[i] = vals_cursor;
        for(int j = 0; j < num_cols_; j++) {
            // printf("i=%d j=%d\n", i, j);
            if(tmp_vals[i*num_cols_+j] > epsilon) {
                // printf("i=%d j=%d vals_cursor=%d\n", i, j, vals_cursor);
                (*vals)[vals_cursor++] = tmp_vals[i*num_cols_+j];
                (*cols)[cols_cursor++] = j;
            }
        }
    }
    (*offsets)[num_rows_] = vals_cursor;

    free(tmp_vals);
}

void MatrixGenerator::generate_dense(int num_rows_, int num_cols_, double** vals){
    *vals = new double[num_rows_ * num_cols_];
    for(int i = 0; i < num_rows_; i++)
        for(int j = 0; j < num_cols_; j++)
            (*vals)[i*num_cols_+j] = ((double)rand()/(double)RAND_MAX);
}


/*********************
Function for HostDenseMat
******************/

HostDenseMat::HostDenseMat(int num_rows_, int num_cols_, double* vals_, bool to_delete_)
        :num_rows(num_rows_), num_cols(num_cols_), vals(vals_), to_delete(to_delete_){
}

HostDenseMat::~HostDenseMat(){
    if(!to_delete) return;
    delete vals;
}

void HostDenseMat::to_device(DeviceDenseMat &d){
    d.num_rows = num_rows;
    d.num_cols = num_cols;

    assert(hipMalloc(&d.vals, num_rows * num_cols * sizeof(double)) == hipSuccess);
    assert(hipMemcpy(d.vals, vals, num_rows * num_cols * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
}

std::ostream& operator<<(std::ostream &os, const HostDenseMat &obj){
    for(int i = 0; i < obj.num_rows; ++i){
        for(int j = 0; j < obj.num_cols; ++j){
            os << std::right << std::setw(6) << std::setprecision(4) << obj.vals[i*obj.num_cols + j] << "\t";
        }
        os << "\n";
    }
    return os;
}

/*********************
Function for DeviceDenseMat
******************/

DeviceDenseMat::~DeviceDenseMat(){
    assert(hipFree(vals) == hipSuccess);
}
void DeviceDenseMat::get_cusparse_descriptor(hipsparseDnMatDescr_t &mat){
    assert(hipsparseCreateDnMat(&mat, num_rows, num_cols, num_cols, vals, HIP_R_64F, HIPSPARSE_ORDER_ROW) == hipSuccess);
}

void DeviceDenseMat::get_cusparse_col_descriptor(hipsparseDnMatDescr_t &mat){
    assert(hipsparseCreateDnMat(&mat, num_rows, num_cols, num_rows, vals, HIP_R_64F, HIPSPARSE_ORDER_COL) == hipSuccess);
}

void DeviceDenseMat::copy_to_host(HostDenseMat &h){
    assert(h.num_rows == num_rows);
    assert(h.num_cols == num_cols);
    assert(hipMemcpy(h.vals, vals, num_rows * num_cols * sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);
}

/*********************
Function for HostSparseMat
******************/

HostSparseMat::HostSparseMat(
            int num_rows_, int num_cols_, int nnz_,
            int *offsets_, int *cols_, double *vals_, bool to_delete_)
    :num_rows(num_rows_), num_cols(num_cols_), nnz(nnz_),
     offsets(offsets_), cols(cols_), vals(vals_), to_delete(to_delete_){
}

HostSparseMat::~HostSparseMat(){
    if(!to_delete) return;

    delete offsets;
    delete cols;
    delete vals;
}

void HostSparseMat::to_dense(HostDenseMat &mat){
    for(int i = 0; i < num_rows; i++)
        for(int j = 0; j < num_cols; j++)
            mat.vals[i*num_cols+j] = 0.0;

    for(int i = 0; i < num_rows; i++) {
        int start_idx = offsets[i];
        int end_idx = offsets[i+1];
        for(int j = start_idx; j < end_idx; j++) {
            int col = cols[j];
            mat.vals[i*num_cols+col] = vals[j];
        }
    }
}

void HostSparseMat::to_device(DeviceSparseMat &d){
    d.num_rows = num_rows;
    d.num_cols = num_cols;
    d.nnz = nnz;

    // malloc
    assert(hipMalloc(&d.offsets, (num_rows+1) * sizeof(int)) == hipSuccess);
    assert(hipMalloc(&d.cols, nnz * sizeof(int)) == hipSuccess);
    assert(hipMalloc(&d.vals, nnz * sizeof(double)) == hipSuccess);

    // copy
    assert(hipMemcpy(d.offsets, offsets, (num_rows+1) * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(d.cols, cols, nnz * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(d.vals, vals, nnz * sizeof(double), hipMemcpyHostToDevice) == hipSuccess);
}

std::ostream& operator<<(std::ostream &os, const HostSparseMat &obj){
    double* tmp = new double[obj.num_rows * obj.num_cols];

    for(int i = 0; i < obj.num_rows; ++i)
        for(int j = 0; j < obj.num_cols; ++j)
            tmp[i*obj.num_cols + j] = 0;

    for(int i = 0; i < obj.num_rows; i++) {
        int start_idx = obj.offsets[i];
        int end_idx = obj.offsets[i+1];
        for(int j = start_idx; j < end_idx; j++) {
            int col = obj.cols[j];
            tmp[i*obj.num_cols+col] = obj.vals[j];
        }
    }

    for(int i = 0; i < obj.num_rows; ++i){
        for(int j = 0; j < obj.num_cols; ++j){
            os << std::right << std::setw(6) << std::setprecision(4) << tmp[i*obj.num_cols + j] << "\t";
        }
        os << "\n";
    }
    return os;
    free(tmp);
}

/*********************
Function for DeviceSparseMat
******************/

DeviceSparseMat::DeviceSparseMat(
        int num_rows_, int num_cols_, int nnz_,
        int *offsets_, int *cols_, double *vals_)
    :num_rows(num_rows_), num_cols(num_cols_), nnz(nnz_),
     offsets(offsets_), cols(cols_), vals(vals_){
}

DeviceSparseMat::~DeviceSparseMat(){
    assert(hipFree(offsets) == hipSuccess);
    assert(hipFree(cols) == hipSuccess);
    assert(hipFree(vals) == hipSuccess);
}

void DeviceSparseMat::get_cusparse_descriptor(
    hipsparseSpMatDescr_t &mat){

    hipsparseCreateCsr(&mat, num_rows, num_cols, nnz,
                      offsets, cols, vals,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
}


void DeviceSparseMat::copy_to_host(HostSparseMat &h){
    assert(h.num_rows == num_rows);
    assert(h.num_cols == num_cols);
    // suppose nnz does not change
    assert(hipMemcpy(h.vals, vals, nnz * sizeof(double), hipMemcpyDeviceToHost) == hipSuccess);
}