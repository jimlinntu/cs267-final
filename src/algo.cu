#include "hip/hip_runtime.h"
#include "../include/algo.cuh"

#define MIN(x, y) (((x) < (y))? (x):(y))

/*********************
Function for CusparseAlgo
******************/

void CusparseAlgo::spmm(
        hipsparseHandle_t &handle,
        hipsparseSpMatDescr_t &S,
        hipsparseDnMatDescr_t &A, hipsparseDnMatDescr_t &C){

    double alpha = 1.0, beta = 0.;
    size_t bufsize = 0;
    hipsparseSpMM_bufferSize(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, S, A, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT,
            &bufsize);
    void *dbuf = NULL;
    assert(hipMalloc(&dbuf, bufsize) == hipSuccess);
    assert(hipsparseSpMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, S, A, &beta, C, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dbuf) == hipSuccess);
    assert(hipFree(dbuf) == hipSuccess);
}

void CusparseAlgo::sddmm(
        hipsparseHandle_t &handle,
        hipsparseSpMatDescr_t &S,
        hipsparseDnMatDescr_t &A){

    
    double alpha = 1.0, beta = 0.;
    size_t bufsize = 0;

    // Get the buffer size
    assert(hipsparseSDDMM_bufferSize(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            &alpha, A, A, &beta, S,
            HIP_R_64F, HIPSPARSE_SDDMM_ALG_DEFAULT, &bufsize) == hipSuccess);

    void *dbuf = NULL;
    assert(hipMalloc(&dbuf, bufsize) == hipSuccess);
    assert(hipsparseSDDMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            &alpha, A, A, &beta, S,
            HIP_R_64F, HIPSPARSE_SDDMM_ALG_DEFAULT, dbuf) == hipSuccess);

    assert(hipFree(dbuf) == hipSuccess);
}

void CusparseAlgo::sddmm(HostSparseMat &S, HostDenseMat &A, HostSparseMat &C){
    // NOTE: S will be modified inplace

    DeviceSparseMat dS;
    DeviceDenseMat dA;

    S.to_device(dS);
    A.to_device(dA);

    hipsparseHandle_t handle = NULL;
    assert(hipsparseCreate(&handle) == hipSuccess);

    hipsparseSpMatDescr_t S_des;
    hipsparseDnMatDescr_t A_des;

    dS.get_cusparse_descriptor(S_des);
    dA.get_cusparse_descriptor(A_des);

    this->sddmm(handle, S_des, A_des);

    // copy the result(modified inplace in dS) back to C
    dS.copy_to_host(C);

    assert(hipsparseDestroySpMat(S_des) == hipSuccess);
    assert(hipsparseDestroyDnMat(A_des) == hipSuccess);
    assert(hipsparseDestroy(handle) == hipSuccess);
}

void CusparseAlgo::sddmm_spmm(
        hipsparseHandle_t &handle,
        hipsparseSpMatDescr_t &C,
        hipsparseDnMatDescr_t &A, 
        hipsparseDnMatDescr_t &B,
        hipsparseDnMatDescr_t &D, 
        hipsparseDnMatDescr_t &E){
    //SDDMM:    (AB) * C    
    double alpha = 1.0, beta = 0.;
    size_t bufsize = 0;
    cusparseConstrainedGeMM_bufferSize(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, A, B, &beta, C, HIP_R_64F,
            &bufsize);
    void *dbuf = NULL;
    assert(hipMalloc(&dbuf, bufsize) == hipSuccess);
    assert(cusparseConstrainedGeMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, A, B, &beta, C, HIP_R_64F, dbuf) == hipSuccess);
    assert(hipFree(dbuf) == hipSuccess);

    //SpMM:     CD = E
    hipsparseSpMM_bufferSize(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, C, D, &beta, E, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT,
            &bufsize);
    // void *dbuf = NULL;
    assert(hipMalloc(&dbuf, bufsize) == hipSuccess);
    assert(hipsparseSpMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, C, D, &beta, E, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dbuf) == hipSuccess);
    assert(hipFree(dbuf) == hipSuccess);
}

/*********************
Function for Algo
******************/

__global__ void spmm_no_shm_kernel(double *A_vals, int *A_cols, int *A_offsets, int A_nnz, double *B_vals, double *C_vals, int A_h, int A_w, int B_h, int B_w) {
    int gy_C = blockIdx.y * 1 + threadIdx.y, gx_C = blockIdx.x * TILE_WIDTH + threadIdx.x;
    
    if(gy_C >= A_h || gx_C >= B_w) return;
    double value = 0.0;
    
    int start_idx = A_offsets[gy_C], end_idx = A_offsets[gy_C+1];

    for(int i = start_idx; i < end_idx; i+=1) {
        int col_A = A_cols[i];
        double val_A = A_vals[i];
        value += val_A * B_vals[col_A*B_w + gx_C];
    }

    C_vals[gy_C*B_w + gx_C] = value;
}

void Algo::spmm_no_shm(HostSparseMat &A, HostDenseMat &B, HostDenseMat &C){
    DeviceSparseMat dA;
    DeviceDenseMat dB, dC;
    A.to_device(dA);
    B.to_device(dB);
    C.to_device(dC);

    int A_h = A.num_rows, A_w = A.num_cols, B_h = B.num_rows, B_w = B.num_cols;
    dim3 dimGrid((B_w+TILE_WIDTH-1)/TILE_WIDTH, A_h);
    dim3 dimBlock(TILE_WIDTH, 1);

    spmm_no_shm_kernel<<<dimGrid, dimBlock>>>(dA.vals, dA.cols, dA.offsets, dA.nnz, dB.vals, dC.vals, A_h, A_w, B_h, B_w);

    dC.copy_to_host(C);
}

__global__ void spmm_kernel(double *A_vals, int *A_cols, int *A_offsets, int A_nnz, double *B_vals, double *C_vals, int A_h, int A_w, int B_h, int B_w) {
    int gy_C = blockIdx.y * 1 + threadIdx.y, gx_C = blockIdx.x * TILE_WIDTH + threadIdx.x;
    
    if(gy_C >= A_h || gx_C >= B_w) return;
    double value = 0.0;

    int lx_C = threadIdx.x;
    
    __shared__ int shm_col_A[TILE_WIDTH];
    __shared__ double shm_val_A[TILE_WIDTH];
    int gx_A_start = A_offsets[gy_C], gx_A_end = A_offsets[gy_C+1];
    int n_steps = (gx_A_end-gx_A_start+TILE_WIDTH-1)/(TILE_WIDTH);

    for(int m = 0; m < n_steps; m++) {
        // m is the tile index
        int start_idx = gx_A_start + m * TILE_WIDTH;

        if(start_idx+lx_C < gx_A_end) {
            shm_col_A[lx_C] = A_cols[start_idx+lx_C];
            shm_val_A[lx_C] = A_vals[start_idx+lx_C];
        } else { // out of range => mark value as zero so it will not be counted
            shm_col_A[lx_C] = 0;
            shm_val_A[lx_C] = 0;
        }
        __syncthreads();

        for(int i = 0; i < TILE_WIDTH; i++) {
            value += shm_val_A[i] * B_vals[shm_col_A[i]*B_w + gx_C];
        }
        __syncthreads();
    }

    C_vals[gy_C*B_w + gx_C] = value;
}

void Algo::spmm(HostSparseMat &A, HostDenseMat &B, HostDenseMat &C){
    DeviceSparseMat dA;
    DeviceDenseMat dB, dC;
    A.to_device(dA);
    B.to_device(dB);
    C.to_device(dC);

    int A_h = A.num_rows, A_w = A.num_cols, B_h = B.num_rows, B_w = B.num_cols;
    dim3 dimGrid((B_w+TILE_WIDTH-1)/TILE_WIDTH, A_h);
    dim3 dimBlock(TILE_WIDTH, 1);

    spmm_kernel<<<dimGrid, dimBlock>>>(dA.vals, dA.cols, dA.offsets, dA.nnz, dB.vals, dC.vals, A_h, A_w, B_h, B_w);

    dC.copy_to_host(C);
}


__global__ void sddmm_shm_kernel(double *S_vals, int *S_cols, double *A_vals, double *C_vals, int *tid_to_vid, int *tid_to_rid, int A_w) {
    int lx = threadIdx.x, gx = blockIdx.x * TILE_WIDTH + lx;

    __shared__ double As[TILE_WIDTH];

    int row_C = tid_to_rid[gx];
    double value = 0.0;

    for(int m = 0; m < (A_w + TILE_WIDTH - 1)/(TILE_WIDTH); m++) {
        if(lx+m*TILE_WIDTH < A_w)
            As[lx] = A_vals[row_C*A_w+lx+m*TILE_WIDTH]; // A_vals[row_C][lx+m*TILE_WIDTH]
        else
            As[lx] = 0; // out of border
        
        __syncthreads();
        if(tid_to_vid[gx] != -1) {
            int n_steps = min(TILE_WIDTH, A_w-m*TILE_WIDTH);
            int col_C = S_cols[tid_to_vid[gx]];
            for(int i = 0; i < n_steps; i++) {
                value += As[i] * A_vals[col_C * A_w + i + m * TILE_WIDTH]; // A_vals[row_C][i+m*TILE_WIDTH] * At_vals[i+m*TILE_WIDTH][col_C]
            }
        }
        __syncthreads();
    }

    if(tid_to_vid[gx] != -1)
        C_vals[tid_to_vid[gx]] = S_vals[tid_to_vid[gx]] * value;
}

__global__ void sddmm_kernel(double *S_vals, int *S_cols, int *S_offsets, int S_nnz, double *A_vals, double *C_vals, int A_h, int A_w) {
    int idx = blockIdx.x * TILE_WIDTH + threadIdx.x;
    if(idx >= S_nnz) return;
    int col_C = S_cols[idx];
    int row_C;

    int i = 0;
    for(; i < A_h; i++) // find where the index sits
        if(S_offsets[i] > idx)
            break;
    row_C = i-1;


    double value = 0.0;
    for(int i = 0; i < A_w; i++)
        value += A_vals[row_C*A_w+i] * A_vals[col_C*A_w+i]; // A_vals[row_C][i] * At_vals[i][col_C] = A_vals[row_C][i] * A_vals[col_C][i]
    C_vals[idx] = S_vals[idx] * value; // C_vals[idx]
}

void Algo::sddmm(HostSparseMat &S, HostDenseMat &A, HostSparseMat &C){

    DeviceSparseMat dS, dC;
    DeviceDenseMat dA;
    S.to_device(dS);
    A.to_device(dA);
    C.to_device(dC);

    int n_threads = 0;
    for(int i = 0; i < S.num_rows; i++) {
        int start_idx = S.offsets[i], end_idx = S.offsets[i+1];
        n_threads += ((end_idx - start_idx + TILE_WIDTH - 1) / TILE_WIDTH) * TILE_WIDTH;
    }

    int *tid_to_vid = new int[n_threads]; // thread id to value id
    int *tid_to_rid = new int[n_threads]; // thread id to row id
    int *tid_to_vid_d;
    int *tid_to_rid_d;

    int k = 0;
    for(int i = 0; i < S.num_rows; i++) {
        int start_idx = S.offsets[i], end_idx = S.offsets[i+1];
        for(int j = start_idx; j < end_idx; j++) {
            tid_to_vid[k] = j;
            tid_to_rid[k] = i;
            k += 1;
        }
        for(int j = end_idx; j < start_idx + ((end_idx - start_idx + TILE_WIDTH - 1) / TILE_WIDTH) * (TILE_WIDTH); j++) {
            tid_to_vid[k] = -1;
            tid_to_rid[k] = i;
            k += 1;
        }
    }

    dim3 dimGrid((n_threads+TILE_WIDTH-1)/TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH);
    hipMalloc(&tid_to_vid_d, sizeof(int) * n_threads);
    hipMalloc(&tid_to_rid_d, sizeof(int) * n_threads);
    hipMemcpy(tid_to_vid_d, tid_to_vid, sizeof(int) * n_threads, hipMemcpyHostToDevice);
    hipMemcpy(tid_to_rid_d, tid_to_rid, sizeof(int) * n_threads, hipMemcpyHostToDevice);

    sddmm_shm_kernel<<<dimGrid, dimBlock>>>(dS.vals, dS.cols, dA.vals, dC.vals, tid_to_vid_d, tid_to_rid_d, dA.num_cols);

    free(tid_to_vid);
    free(tid_to_rid);
    hipFree(tid_to_vid_d);
    hipFree(tid_to_rid_d);
    dC.copy_to_host(C);

}

void Algo::sddmm_block_over_nnz(HostSparseMat &S, HostDenseMat &A, HostSparseMat &C){
    DeviceSparseMat dS, dC;
    DeviceDenseMat dA;
    S.to_device(dS);
    A.to_device(dA);
    C.to_device(dC);

    int A_h = A.num_rows, A_w = A.num_cols;
    int nnz = S.nnz;
    dim3 dimGrid((nnz+TILE_WIDTH-1)/TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH);

    sddmm_kernel<<<dimGrid, dimBlock>>>(dS.vals, dS.cols, dS.offsets, dS.nnz, dA.vals, dC.vals, A_h, A_w);

    dC.copy_to_host(C);
}

__global__ void count_num_blocks_in_each_row(
        int S_num_rows, int *S_offsets, int *block_offsets){

    int row_id = blockDim.x * blockIdx.x + threadIdx.x;
    if(row_id >= S_num_rows) return;

    int nnz_col = S_offsets[row_id+1] - S_offsets[row_id];
    // += the number of blocks needed for this row
    atomicAdd(&block_offsets[row_id+1], (nnz_col + TILE_WIDTH - 1) / TILE_WIDTH);
}

__global__ void sddmm_block_over_nnz_in_same_row_kernel(
    int S_num_rows, int *S_offsets, int *S_cols, double *S_vals,
    int A_num_cols, double *A_vals,
    double *C_vals,
    int *block_offsets){

    // Each block must first search which row is belongs to by binary search
    int block_idx = blockIdx.x;
    int l = 0, r = S_num_rows;
    int mid;

    __shared__ int shm_row_idx;
    // only one thread needs to compute this
    if(threadIdx.x == 0){
        // l = upperbound(block_offsets, block_idx)
        // find the smallest(first) idx s.t. block_offsets[idx] > block_idx
        while(l < r){
            mid = (l + r) / 2;
            if(block_offsets[mid] <= block_idx){
                l = mid+1;
            }else{
                r = mid;
            }
        }
        assert(l <= S_num_rows);
        shm_row_idx = l-1;
    }
    __syncthreads();

    int row_idx = shm_row_idx; // copy to this thread's private space
    int start = S_offsets[row_idx], end = S_offsets[row_idx+1];

    int _j = (blockIdx.x - block_offsets[row_idx]) * TILE_WIDTH + threadIdx.x;
    int j = -1;
    if(start + _j < end) j = S_cols[start + _j];

    __shared__ double A_shm[TILE_WIDTH];

    double value = 0.;
    for(int k = 0; k < A_num_cols; k += TILE_WIDTH){
        int my_k = k + threadIdx.x;
        if(my_k < A_num_cols){
            A_shm[threadIdx.x] = A_vals[row_idx * A_num_cols + my_k];
        }
        __syncthreads();

        const int bound_tile_width = MIN(TILE_WIDTH, A_num_cols - k);

        if(j != -1){
            for(int kk = 0; kk < bound_tile_width; ++kk){
                value += A_shm[kk] * A_vals[j * A_num_cols + k + kk];
            }
        }
        __syncthreads();
    }

    // Write back
    if(j != -1) C_vals[start + _j] = S_vals[start + _j] * value;
}
void Algo::sddmm_block_over_nnz_but_in_same_row(HostSparseMat &S, HostDenseMat &A, HostSparseMat &C){
    DeviceSparseMat dS, dC;
    DeviceDenseMat dA;

    S.to_device(dS);
    A.to_device(dA);
    C.to_device(dC);

    // block_offsets[row_id] = # of blocks needed this row
    int *block_offsets;

    assert(hipMalloc(&block_offsets, sizeof(int) * (S.num_rows+1)) == hipSuccess);
    // set 0 initially
    assert(hipMemset(block_offsets, 0, sizeof(int) * (S.num_rows+1)) == hipSuccess);

    const int num_threads = 256;
    // Parallelize over # of rows
    count_num_blocks_in_each_row<<<(S.num_rows + num_threads - 1)/ num_threads, num_threads>>>(
                S.num_rows, dS.offsets, block_offsets);
    // prefix sum
    // block_offsets[i] = # of blocks that are in [0, i) rows
    thrust::device_ptr<int> ptr(block_offsets);
    thrust::inclusive_scan(ptr+1, ptr+S.num_rows+1, ptr+1);

    int num_blocks = 0;

    // Only copy the total number of blocks back
    hipMemcpy(&num_blocks, block_offsets + S.num_rows, sizeof(int), hipMemcpyDeviceToHost);

    sddmm_block_over_nnz_in_same_row_kernel<<<num_blocks, TILE_WIDTH>>>(
            S.num_rows, dS.offsets, dS.cols, dS.vals,
            A.num_cols, dA.vals,
            dC.vals, block_offsets);

    dC.copy_to_host(C);

    assert(hipFree(block_offsets) == hipSuccess);
}


__global__ void sddmm_launch_kernel_as_dense_matrix_kernel(
        int S_num_rows, int *S_offsets, int *S_cols, double *S_vals,
        int A_num_cols, double *A_vals,
        double *C_vals){

    int i = blockIdx.x;
    int _j_first = blockIdx.y * blockDim.y;
    int _j = blockIdx.y * blockDim.y + threadIdx.y;

    int start = S_offsets[i], end = S_offsets[i+1];
    // if the first thread in this block has nothing to do,
    // this block has no work to do
    if(start + _j_first >= end) return;

    int j = (start + _j < end)? (S_cols[start + _j]):(-1);

    __shared__ double A_shm[TILE_WIDTH];

    double value = 0.;
    for(int k = 0; k < A_num_cols; k += TILE_WIDTH){
        int my_k = k + threadIdx.y;
        if(my_k < A_num_cols){
            A_shm[threadIdx.y] = A_vals[i * A_num_cols + my_k];
        }
        __syncthreads();

        const int bound_tile_width = MIN(TILE_WIDTH, A_num_cols - k);

        if(j != -1){
            for(int kk = 0; kk < bound_tile_width; ++kk){
                value += A_shm[kk] * A_vals[j * A_num_cols + (k + kk)];
            }
        }
        __syncthreads();
    }

    // Write to C
    if(j != -1) C_vals[start + _j] = S_vals[start + _j] * value;
}

void Algo::sddmm_launch_kernel_as_dense_matrix(
        HostSparseMat &S, HostDenseMat &A, HostSparseMat &C){

    DeviceSparseMat dS, dC;
    DeviceDenseMat dA;

    S.to_device(dS);
    A.to_device(dA);
    C.to_device(dC);

    // Launch the kernel as if it is a dense matrix
    dim3 threadsPerBlock(1, TILE_WIDTH);
    dim3 numBlocks(S.num_rows, (S.num_cols + TILE_WIDTH - 1) / TILE_WIDTH);

    sddmm_launch_kernel_as_dense_matrix_kernel<<<numBlocks, threadsPerBlock>>>(
        S.num_rows, dS.offsets, dS.cols, dS.vals,
        A.num_cols, dA.vals,
        dC.vals);

    dC.copy_to_host(C);
}

__global__ void sddmm_block_over_nnz_if_same_row_use_shm_kernel(
        int S_num_rows, int S_nnz, int *S_offsets, int *S_cols, double *S_vals,
        int A_num_cols, double *A_vals,
        double *C_vals){

    int _j = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int bound = MIN(TILE_WIDTH, S_nnz - blockIdx.x * TILE_WIDTH);

    __shared__ int row_indices[TILE_WIDTH];
    __shared__ double A_shm[TILE_WIDTH];

    // find this element's row idx (i.e. i)
    int l = 0, r = S_num_rows;
    int mid;

    // Binary search to find the row idx
    if(threadIdx.x < bound){
        while(l < r){
            mid = (l+r)/2;
            if(S_offsets[mid] <= _j){
                l = mid+1;
            }else{
                r = mid;
            }
        }
        assert(l <= S_num_rows);
        row_indices[threadIdx.x] = l-1;
    }
    __syncthreads();

    double value = 0.;
    // If the first row_idx and the last row_idx are the same,
    // we can use shared memory (and all threads will enter this branch)
    if(row_indices[0] == row_indices[bound-1]){
        int i = row_indices[0]; // load from the shared mem
        int j = (threadIdx.x < bound)?(S_cols[_j]):(-1);

        for(int k = 0; k < A_num_cols; k += TILE_WIDTH){
            int my_k = k + threadIdx.x;
            if(my_k < A_num_cols){
                A_shm[threadIdx.x] = A_vals[i * A_num_cols + my_k];
            }
            __syncthreads();
            if(j != -1){
                const int bound_tile_width = MIN(TILE_WIDTH, A_num_cols - k);
                for(int kk = 0; kk < bound_tile_width; ++kk){
                    value += A_shm[kk] * A_vals[j * A_num_cols + (k + kk)];
                }
            }
            __syncthreads();
        }
    }else if(threadIdx.x < bound){
        // Otherwise, we cannot use shared memory to accelerate
        // in this case, each thread will compute by its own element without collaborating
        int i = l-1;
        int j = S_cols[_j];
        for(int k = 0; k < A_num_cols; ++k){
            value += A_vals[i * A_num_cols + k] * A_vals[j * A_num_cols + k];
        }
    }
    // Write to C
    if(threadIdx.x < bound) C_vals[_j] = S_vals[_j] * value;
}

void Algo::sddmm_block_over_nnz_if_same_row_use_shm(
        HostSparseMat &S, HostDenseMat &A, HostSparseMat &C){

    DeviceSparseMat dS, dC;
    DeviceDenseMat dA;

    S.to_device(dS);
    A.to_device(dA);
    C.to_device(dC);

    dim3 threadsPerBlock(TILE_WIDTH);
    dim3 numBlocks((S.nnz + TILE_WIDTH - 1) / TILE_WIDTH);

    sddmm_block_over_nnz_if_same_row_use_shm_kernel<<<numBlocks, threadsPerBlock>>>(
        S.num_rows, S.nnz, dS.offsets, dS.cols, dS.vals,
        A.num_cols, dA.vals,
        dC.vals);

    dC.copy_to_host(C);
}

void Algo::sddmm_seq(HostSparseMat &S, HostDenseMat &A, HostSparseMat &C){
    for(int i = 0; i < C.num_rows; i++){
        int row_C = i;
        int start_idx = C.offsets[i], end_idx = C.offsets[i+1];
        for(int j = start_idx; j < end_idx; j++) {
            int col_C = C.cols[j];
            double value = 0.0;
            for(int k = 0; k < A.num_cols; k++)
                value += A[row_C*A.num_cols+k] * A[col_C*A.num_cols+k];
            C.vals[j] = value * S.vals[j];
        }
    }
}

void Algo::sddmm_spmm(){
}

void Algo::ddmm_seq(HostDenseMat &A, HostDenseMat &B, HostDenseMat &C){
    int A_num_rows = A.num_rows, A_num_cols = A.num_cols;
    int B_num_rows = B.num_rows, B_num_cols = B.num_cols;
    assert(A_num_cols == B_num_rows);

    for(int i = 0; i < A_num_rows; i++) {
        for(int j = 0; j < B_num_cols; j++) {
            C[i*B_num_cols+j] = 0.0;
            for(int k = 0; k < A_num_cols; k++) {
                C[i*B_num_cols+j] += A[i*A_num_cols+k] * B[k*B_num_cols+j];
            }
        }
    }
}
